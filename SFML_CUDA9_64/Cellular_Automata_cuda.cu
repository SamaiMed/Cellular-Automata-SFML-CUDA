#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include "SFML\Graphics.hpp"
#define N  64


int onedim(int n, int i, int j) { return n*i + j; }

void Update(int w, int h, int *map, int *stats) {
	for (int i = 1; i < w - 1; i++) {
		for (int j = 1; j < h - 1; j++) {

			int a = map[onedim(h, i - 1, j + 1)] + map[onedim(h, i, j + 1)] + map[onedim(h, i + 1, j + 1)] +
				map[onedim(h, i - 1, j)] + 0 + map[onedim(h, i + 1, j)] +
				map[onedim(h, i - 1, j - 1)] + map[onedim(h, i, j - 1)] + map[onedim(h, i + 1, j - 1)];

			if (map[onedim(h, i, j)] == 1) {
				stats[onedim(h, i, j)] = (int)((a == 2 || a == 3) ? 1 : 0);
			}
			else {
				stats[onedim(h, i, j)] = (int)(a == 3 ? 1 : 0);
			}
		}
	}	
}

__global__ void Update_CUDA(int w, int h, int *map, int *stats) {

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int stride_x = gridDim.x * blockDim.x;
	int stride_y = gridDim.y * blockDim.y;
	if (row > 0 && col > 0 && row < w - 1 && col < h - 1) {
		for (int i = row; i < w - 1; i += stride_x) {
			for (int j = col; j < h - 1; j += stride_y) {

				int a = map[h*(i - 1)+j + 1] + map[h*i+j + 1] + map[h*(i + 1)+j + 1] +
					map[h*(i - 1)+j] + 0 + map[h*(i + 1)+j] +
					map[h*(i - 1)+j - 1] + map[h*i+j - 1] + map[h*(i + 1)+j - 1];

				if (map[h*i+j] == 1) {
					stats[h*i+j] = (int)((a == 2 || a == 3) ? 1 : 0);
				}
				else {
					stats[h*i+j] = (int)(a == 3 ? 1 : 0);
				}
			}
		}
	}

	
}


int main() {
	const int w = 1300;
	const int h = 766;
	const float wf = (float)w;
	const float hf = (float)h;
	int *map, *stats;
	sf::Uint8 *pixels=new sf::Uint8[w * h * 4];
	sf::RenderWindow window(sf::VideoMode(w, h), "SFML works!");

	sf::Clock clock;
	sf::Time time;

	sf::RectangleShape player(sf::Vector2f(wf, hf));
	player.setPosition(0.f, 0.f);
	sf::Texture player_texture;
	sf::Image image;
	sf::Image *imo;
	image.create(w, h);
	
	hipMallocManaged(&map, w*h*sizeof(int));
	hipMallocManaged(&stats, w*h*sizeof(int));
	


	dim3 threads_per_block(32, 32, 1); // A 16 x 16 block threads
	dim3 number_of_blocks((w / threads_per_block.x) + 1, (h / threads_per_block.y) + 1, 1);

	for (int i = 0; i < w; i++) {
		for (int j = 0; j < h; j++) {
			map[onedim(h,i,j)] = 0;
			if (((float)rand() / (RAND_MAX + 1.0)) < 0.75f) {
				image.setPixel(i, j, sf::Color::Blue);
				map[onedim(h,i,j)] = 1;
			}
		}
	}
	player_texture.loadFromImage(image);
	player.setTexture(&player_texture);

	while (window.isOpen())
	{
		time = clock.getElapsedTime();
		clock.restart().asSeconds();
		float fps = 1.0f / time.asSeconds();

		printf("FPS:: %f \r", fps);
		sf::Event event;

		while (window.pollEvent(event))
		{

			switch (event.type)
			{
			case sf::Event::Closed:
				window.close();
				break;
			case sf::Event::Resized:
				printf("Windows size : %d, %d \n", event.size.height, event.size.width);
				//window.setView(sf::View(sf::FloatRect(0, 0, event.size.width, event.size.height)));
				break;
			case sf::Event::TextEntered:
				if (event.text.unicode < 128) {
					printf("%c", event.text.unicode);
				}
				break;
			}


		}
	//	Update(w, h, map,stats, &image); // for test 
		Update_CUDA << < number_of_blocks, threads_per_block >> > (w, h, map, stats);
		hipDeviceSynchronize();

		for (int i = 0; i < w; i++) {
			for (int j = 0; j < h; j++) {
				map[onedim(h, i, j)] = stats[onedim(h, i, j)];
				if (map[onedim(h, i, j)] == 1) {
					image.setPixel(i, j, sf::Color::Blue);
				}
				else {
					image.setPixel(i, j, sf::Color::Black);
				}
			}
		}

		player_texture.loadFromImage(image);
		player.setTexture(&player_texture);
		window.clear();
		window.draw(player);
		window.display();
	}

	hipFree(map); hipFree(stats);
	return 0;
}